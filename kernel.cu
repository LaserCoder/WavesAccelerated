#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/async/copy.h>
#include <thrust/async/reduce.h>
#include <thrust/functional.h>
#include <thrust/complex.h>
#include <thrust/random.h>
#include <numeric>
#include <hip/hip_complex.h>

typedef thrust::complex<double> complex;

__global__ void myKernel(   int dim,
                            int iterations,
                            int nSamples,
                            complex* Ep,
                            complex* Em,
                            complex* Pp,
                            complex* Pm,
                            complex* dEpdz,
                            complex* dEmdz,
                            complex* dEp2dz2,
                            complex* dEm2dz2,
                            complex* dEpt0,
                            complex* dEmt0,
                            complex* dEpdt,
                            complex* dEmdt,
                            complex* tmpdz2,
                            complex* tmpdz,
                            double* power
                        )
{
    double J  = 1100;
    double R1 = 0.09;
    double R2 = 1;
    double c = 299792458;
    double n = 3.3;
    double kpp = -2.000000000000000e-24;
    double aw = 400;
    double gammaK = 0;
    double g0 = 819.4299;
    double gc = 1;
    double T1 = 4.0000e-13;
    double T2 = 5.0000e-14;
    double dz =  8.0000e-07;
    double dt = 8.8061e-15;
    double Psat = 8.189848267404146e+12;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Initialize Ep and Em
    if(tid < dim) {
        Ep[tid] = cos(2*3.14*2*tid*tid/dim/dim);
        Em[tid] = 0;
    }

    for(int i = 0; i < iterations ; i++)
    {
        // Calculate Power
        if(tid < dim)
        {
            Pp[tid] = abs(Ep[tid]) * abs(Ep[tid]);
            Em[tid] = abs(Em[tid]) * abs(Em[tid]);
        }
        // Calculate dEpdz
        if(tid == 0)
        {
            tmpdz[tid] = Em[tid+1]*sqrt(R1);

        }
        else if(tid < dim + 1)
        {
            tmpdz[tid]  = Ep[tid-1];
        }
        __syncthreads();
        dEpdz[tid] = (tmpdz[tid+1] - tmpdz[tid])/ dz;
        //calculate dEmdz
        
    }
    
}

std::vector<double> wrapper(int dim)
{
    int threadsPerBlock = 128;
    int blocksPerGrid = 32; 
    dim = threadsPerBlock*blocksPerGrid - 2;
    complex* Ep;
    complex* Em;
    complex* Pp;
    complex* Pm;
    complex* dEpdz;
    complex* dEmdz;
    complex* dEp2dz2;
    complex* dEm2dz2;
    complex* dEpt0;
    complex* dEmt0;
    complex* dEpdt;
    complex* dEmdt;
    complex* tmpdz2;
    complex* tmpdz;
    double* power;
    size_t size = dim * sizeof(complex);
    hipMalloc(&Ep, size);
    hipMalloc(&Em, size);
    hipMalloc(&Pp, size);
    hipMalloc(&Pm, size);
    hipMalloc(&dEpdz, size);
    hipMalloc(&dEmdz, size);
    hipMalloc(&dEp2dz2, size);
    hipMalloc(&dEm2dz2, size);
    hipMalloc(&dEpt0, size);
    hipMalloc(&dEmt0, size);
    hipMalloc(&dEpdt, size);
    hipMalloc(&dEmdt, size);
    hipMalloc(&tmpdz2, size);
    hipMalloc(&tmpdz, size);
    hipMalloc(&power, 5'000*sizeof(double));
    myKernel<<<blocksPerGrid, threadsPerBlock>>>(Ep, Em, Pp, Pm, dEpdz, dEmdz, dEp2dz2, dEm2dz2, dEpt0, dEmt0, dEpdt, dEmdt, tmpdz2, tmpdz, power);
    std::vector<double> power_host(5000);
    hipError_t err = hipMemcpy(power_host.data(), power, 5000 * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(Ep);
    hipFree(Em);
    hipFree(Pp);
    hipFree(Pm);
    hipFree(dEpdz);
    hipFree(dEmdz);
    hipFree(dEp2dz2);
    hipFree(dEm2dz2);
    hipFree(dEpt0);
    hipFree(dEmt0);
    hipFree(dEpdt);
    hipFree(dEmdt);
    hipFree(tmpdz2);
    hipFree(tmpdz);
    hipFree(power);
    return power_host;
}